
#include <hip/hip_runtime.h>
extern "C"
__global__ void grav(int n, double G,
	double *mass, double *posX, double *posY,
	double *rForceX, double *rForceY)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    if (i < n && j < n && i != j)
    {
        double relX = posX[j] - posX[i];
        double relY = posY[j] - posY[i];
        
        double dist2 = relX * relX + relY * relY;
        
        double scl = G * mass[i] * mass[j] * rsqrt(dist2 * dist2 * dist2);
        
        rForceX[i] += relX * scl;
        rForceY[i] += relY * scl;
    }

}
