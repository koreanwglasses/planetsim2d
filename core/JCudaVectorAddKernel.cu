
#include <hip/hip_runtime.h>
extern "C"
__global__ void add(int n, float *ax, float *ay, float *bx, float *by, float *sumx, float *sumy)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i<n)
    {
        sumx[i] = ax[i] + bx[i];
        sumy[i] = ay[i] + by[i];
    }

}
